#include "hip/hip_runtime.h"
// Copyright 2021-2025 Alfredo A. Correa
// Distributed under the Boost Software License, Version 1.0.
// https://www.boost.org/LICENSE_1_0.txt

#include <boost/core/lightweight_test.hpp>

#include <boost/multi/adaptors/thrust.hpp>
#include <boost/multi/adaptors/thrust/managed_allocator.hpp>
#include <boost/multi/array.hpp>

#include <hip/hip_runtime_api.h>
#include <thrust/complex.h>
#include <thrust/device_allocator.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/memory.h>
#include <thrust/uninitialized_copy.h>
#include <thrust/universal_allocator.h>

#include <boost/timer/timer.hpp>

#include <numeric>

namespace multi = boost::multi;

#ifdef __NVCC__
template<>
inline constexpr bool ::boost::multi::force_element_trivial_default_construction<::std::complex<double>> = true;
template<>
inline constexpr bool ::boost::multi::force_element_trivial_default_construction<::std::complex<float>> = true;
template<>
inline constexpr bool ::boost::multi::force_element_trivial_default_construction<::thrust::complex<double>> = true;
template<>
inline constexpr bool ::boost::multi::force_element_trivial_default_construction<::thrust::complex<float>> = true;
#else  // vvv nvcc (12.1?) doesn't support this kind of customization: "error: expected initializer before ‘<’"
template<class T>
inline constexpr bool ::boost::multi::force_element_trivial_default_construction<::std::complex<T>> = std::is_trivially_default_constructible<T>::value;
template<class T>
inline constexpr bool ::boost::multi::force_element_trivial_default_construction<::thrust::complex<T>> = std::is_trivially_default_constructible<T>::value;
#endif

namespace {

template<class T> using test_allocator =
	//  multi::thrust::cuda::managed_allocator<T>
	thrust::cuda::allocator<T>;
}

auto universal_memory_supported() -> bool {
	std::cout << "testing for universal memory supported" << std::endl;
	int d;
	hipGetDevice(&d);
	int is_cma = 0;
	hipDeviceGetAttribute(&is_cma, hipDeviceAttributeConcurrentManagedAccess, d);
	if(is_cma) {
		std::cout << "universal memory is supported" << std::endl;
	} else {
		std::cout << "universal memory is NOT supported" << std::endl;
	}
	return (is_cma == 1)?true:false;
}

template<class T, class Ptr>
struct ::thrust::iterator_system<multi::device_array_iterator<T, Ptr> > {
	using type = typename ::thrust::iterator_system<Ptr>::type;
};

auto main() -> int {  // NOLINT(readability-function-cognitive-complexity,bugprone-exception-escape)
	
	// BOOST_AUTO_TEST_CASE(cuda_allocators)
	{

		multi::array<double, 1, thrust::cuda::allocator<double>> A1(200, 0.0);

		BOOST_TEST( size(A1) == 200 );
		A1[100] = 1.0;

		multi::array<double, 1, thrust::cuda::allocator<double>> const B1(200, 2.0);
		BOOST_TEST( B1[10] == 2.0 );

		A1[10] = B1[10];
		BOOST_TEST( A1[10] == 2.0 );
	}
	std::cout << "line " << __LINE__ << std::endl;

	// BOOST_AUTO_TEST_CASE(cuda_universal_empty)
	if(universal_memory_supported())
	{
		using complex = thrust::complex<double>;
		multi::array<complex, 2, thrust::cuda::universal_allocator<complex>> A;
		multi::array<complex, 2, thrust::cuda::universal_allocator<complex>> B = A;
		BOOST_TEST( A.is_empty() );
		BOOST_TEST( B.is_empty() );
		BOOST_TEST( A == B );
	}

	std::cout << "line " << __LINE__ << std::endl;

	// BOOST_AUTO_TEST_CASE(cuda_1d_initlist)
	{
		multi::array<double, 1, thrust::device_allocator<double>> A1 = {1.0, 2.0, 3.0};
		BOOST_TEST( A1.size() == 3 );

		// BOOST_TEST( size(A1) == 200 );
		// A1[100] = 1.0;

		// multi::array<double, 1, thrust::cuda::allocator<double>> const B1(200, 2.0);
		// BOOST_TEST( B1[10] == 2.0 );

		// A1[10] = B1[10];
		// BOOST_TEST( A1[10] == 2.0 );

		std::cout << "line " << __LINE__ << std::endl;

		
		{
			thrust::device_vector<int, thrust::device_allocator<int> > vA(3, 44);
			thrust::device_vector<int, thrust::device_allocator<int> > vB(3, 0);

			std::cout << "line " << __LINE__ << std::endl;

			thrust::copy(thrust::cuda::par, vA.begin(), vA.end(), vB.begin());

			static_assert( std::is_same_v<decltype(vA.data()), thrust::device_ptr<int>> );

			std::cout << "line " << __LINE__ << std::endl;

			thrust::copy(thrust::cuda::par, vA.data(), vA.data() + vA.size(), vB.begin());

			BOOST_TEST( vB[0] == 44 );
		}
		{
			thrust::device_allocator<int> devalloc;

			std::cerr << "line " << __LINE__ << std::endl;

			auto Aptr = devalloc.allocate(3);

			std::cerr << "line " << __LINE__ << std::endl;

			thrust::fill(thrust::cuda::par , Aptr, Aptr + 3, 44);

			std::cerr << "line " << __LINE__ << std::endl;

			auto Bptr = devalloc.allocate(3);

			std::cerr << "line " << __LINE__ << std::endl;

			thrust::fill(thrust::cuda::par , Bptr, Bptr + 3, 0);

			std::cerr << "line " << __LINE__ << std::endl;

			thrust::copy(thrust::cuda::par, Aptr, Aptr + 3, Bptr);

			devalloc.deallocate(Aptr, 3);
			devalloc.deallocate(Bptr, 3);
		}
		{
			thrust::device_allocator<int> devalloc;

			auto Aptr = devalloc.allocate(3);

			std::cerr << "line " << __LINE__ << std::endl;

			thrust::uninitialized_fill(thrust::cuda::par , Aptr, Aptr + 3, 44);

			std::cerr << "line " << __LINE__ << std::endl;

			auto Bptr = devalloc.allocate(3);

			std::cerr << "line " << __LINE__ << std::endl;

			thrust::uninitialized_fill(thrust::cuda::par , Bptr, Bptr + 3, 44);

			std::cerr << "line " << __LINE__ << std::endl;

			multi::array_ref<int, 1, thrust::device_ptr<int> > aA(Aptr, 3);

			std::cout << "line " << __LINE__ << std::endl;

			multi::array_ref<int, 1, thrust::device_ptr<int> > aB(Bptr, 3);

			std::cout << "line " << __LINE__ << std::endl;

			thrust::copy_n(thrust::cuda::par, Aptr, aA.size(), Bptr);

			std::cout << "line " << __LINE__ << std::endl;

			BOOST_TEST( Aptr == aA.data_elements() );
			BOOST_TEST( Bptr == aB.data_elements() );

			std::cout << "line " << __LINE__ << std::endl;

			thrust::copy_n(thrust::cuda::par, aA.data_elements(), aA.size(), aB.data_elements());

			std::cout << "line " << __LINE__ << std::endl;

			auto Ait2 = multi::array_iterator<int, 1, thrust::device_ptr<int> >(Aptr, 1);
			auto Ait = multi::device_array_iterator<int, ::thrust::device_ptr<int> >(Aptr, 1);

			std::cout << "line " << __LINE__ << std::endl;

			static_assert( std::is_same_v< 
				::thrust::iterator_system<decltype(Ait)>::type,
				::thrust::iterator_system< thrust::device_ptr<int> >::type
			>);

			std::cout << "line " << __LINE__ << std::endl;

			thrust::copy_n(Ait, aA.size(), aB.data_elements());

			std::cout << "line " << __LINE__ << std::endl;

			thrust::copy_n(thrust::cuda::par, Ait, aA.size(), aB.data_elements());

			std::cout << "line " << __LINE__ << std::endl;

			thrust::copy_n(thrust::cuda::par, Ait2, aA.size(), aB.data_elements());

			std::cout << "line " << __LINE__ << std::endl;

			thrust::copy_n(thrust::cuda::par, aA.begin(), aA.size(), aB.data_elements());

			std::cout << "line " << __LINE__ << std::endl;

			thrust::copy_n(thrust::cuda::par, aA.data_elements(), aA.size(), aB.begin());

			std::cout << "line " << __LINE__ << std::endl;

			static_assert( std::is_same_v<decltype(aA.data_elements()), thrust::device_ptr<int> > );

			thrust::copy(thrust::cuda::par, aA.data_elements(), aA.data_elements() + aA.size(), aB.begin());

			std::cout << "line " << __LINE__ << std::endl;

			thrust::copy(thrust::cuda::par, aA.begin(), aA.end(), aB.begin());

			BOOST_TEST( aB[0] == 44 );
		}

		std::cout << "line " << __LINE__ << std::endl;

		{
			multi::array<int, 1, thrust::device_allocator<int> > A(3, 44);
			multi::array<int, 1, thrust::device_allocator<int> > B(3, 0);

			std::cout << "line " << __LINE__ << std::endl;

			BOOST_TEST( A[1] == 44 );

			std::cout << "line " << __LINE__ << std::endl;

			thrust::transform(A.data_elements(), A.data_elements() + A.num_elements(), B.data_elements(), [] __device__(int const& elem) -> int { return elem * 2; });

			std::cout << "line " << __LINE__ << std::endl;

			B[0] = A[0]*2;

			std::cout << "line " << __LINE__ << std::endl;

			thrust::copy(A.begin(), A.end(), B.begin());

			std::cout << "line " << __LINE__ << std::endl;

			thrust::transform(A.begin(), A.end(), B.begin(), [] __device__(int const& elem) -> int { return elem * 2; });

			std::cout << "line " << __LINE__ << std::endl;
			int B1 = B[1];
			BOOST_TEST( B1 == 88 );

			std::cout << "line " << __LINE__ << std::endl;

			BOOST_TEST( B[1] == 88 );
		}
		
		{
			multi::array<int, 1, thrust::device_allocator<int> > A = {1, 2, 3};
			multi::array<int, 1, thrust::device_allocator<int> > B(3, 0);

			BOOST_TEST( A[1] == 2 );

			thrust::transform(A.begin(), A.end(), B.begin(), [] __device__(int elem) { return elem * 2; });

			std::cout << "line " << __LINE__ << std::endl;
			int B1 = B[1];
			BOOST_TEST( B1 == 4 );

			std::cout << "line " << __LINE__ << std::endl;
			BOOST_TEST( B[1] == 4 );
		}

		std::cout << "line " << __LINE__ << std::endl;

		{
			multi::array<double, 1, thrust::device_allocator<double>> A = {1.0, 2.0, 3.0};
			multi::array<double, 1, thrust::device_allocator<double>> B(3);

			// // for(int i = 0; i != A.size(); ++i) { B[i] = A[i]*2.0; }
			// // for(auto i : A.extension()) { B[i] = A[i]*2.0; }

			thrust::transform(A.begin(), A.end(), B.begin(), [] __device__(double const& elem) { return elem * 2.0; });

			BOOST_TEST( B[1] == 4.0 );
		}
	}

	std::cout << "line " << __LINE__ << std::endl;

	// BOOST_AUTO_TEST_CASE(test_univ_alloc)
	if(universal_memory_supported())
	{
		multi::array<double, 2, thrust::cuda::universal_allocator<double>> Dev({128, 128});
		*raw_pointer_cast(Dev.base()) = 99.0;
	}

	std::cout << "line " << __LINE__ << std::endl;

	// BOOST_AUTO_TEST_CASE(mtc_universal_array)
	if(universal_memory_supported())
	{
		multi::thrust::cuda::universal_array<double, 2> Dev({128, 128});
		*raw_pointer_cast(Dev.base()) = 99.0;
	}

	std::cout << "line " << __LINE__ << std::endl;

	// BOOST_AUTO_TEST_CASE(mtc_universal_coloncolon_array)
	if(universal_memory_supported())
	{
		multi::thrust::cuda::universal::array<double, 2> Dev({128, 128});
		*raw_pointer_cast(Dev.base()) = 99.0;
	}

	std::cout << "line " << __LINE__ << std::endl;

	// BOOST_AUTO_TEST_CASE(test_alloc)
	{
		multi::array<double, 2, thrust::cuda::allocator<double>> Dev({128, 128});
		// *raw_pointer_cast(Dev.base()) = 99.0;  // segmentation fault (correct behavior)
	}

#ifdef NDEBUG

	std::cout << "line " << __LINE__ << std::endl;

	auto const n = 1024;

	// BOOST_AUTO_TEST_CASE(thrust_copy_1D_issue123_double)
	{  // BOOST_AUTO_TEST_CASE(fdfdfdsfds) { using T = char;
		using T = double;

		static_assert(std::is_trivially_default_constructible_v<T>);
		static_assert(std::is_trivially_copy_constructible_v<T>);
		static_assert(std::is_trivially_assignable_v<T&, T>);

		multi::array<T, 1, test_allocator<T>> Devc(multi::extensions_t<1>{n * n});
		multi::array<T, 1, test_allocator<T>> Dev2(multi::extensions_t<1>{n * n});
		multi::array<T, 1>                    Host(multi::extensions_t<1>{n * n});
		std::iota(Host.elements().begin(), Host.elements().end(), 12.0);
		multi::array<T, 1> Hos2(multi::extensions_t<1>{n * n});

		std::cout << "| 1D `" << typeid(T).name() << "` total data size: " << Host.num_elements() * sizeof(T) / 1073741824. << " GB | speed |\n|---|---|" << std::endl;
		{
			Devc = Host;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc = Host;
			hipDeviceSynchronize();
			std::cout << "| contiguous host -> devc | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc.sliced(0, n * n / 2) = Host.sliced(0, n * n / 2);
			hipDeviceSynchronize();
			std::cout << "| sliced     host -> devc | " << Host.sliced(0, n * n / 2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc.strided(2) = Host.strided(2);
			hipDeviceSynchronize();
			std::cout << "| strided    host -> devc | " << Host.strided(2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2 = Devc;
			hipDeviceSynchronize();
			std::cout << "| contiguous devc -> host | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Hos2 == Host );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.sliced(0, n * n / 2) = Devc.sliced(0, n * n / 2);
			hipDeviceSynchronize();
			std::cout << "| sliced     devc -> host | " << Host.sliced(0, n * n / 2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Hos2 == Host );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.strided(2) = Devc.strided(2);
			hipDeviceSynchronize();
			std::cout << "| strided    devc -> host | " << Host.strided(2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Hos2 == Host );
		}

		std::cout << "line " << __LINE__ << std::endl;

		{
			boost::timer::auto_cpu_timer t{""};
			Dev2 = Devc;
			hipDeviceSynchronize();
			std::cout << "| contiguous devc -> devc | " << Dev2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			auto                         Dev3 = Devc;
			hipDeviceSynchronize();
			std::cout << "| copy_ctr   devc -> devc | " << Devc.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev3 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			hipMemcpy(raw_pointer_cast(Dev2.data_elements()), raw_pointer_cast(Devc.data_elements()), Devc.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();
			std::cout << "| hipMemcpy devc -> devc | " << Dev2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			hipMemcpy(raw_pointer_cast(Dev2.data_elements()), raw_pointer_cast(Devc.data_elements()), Devc.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();
			std::cout << "| hipMemcpy devc -> devc | " << Dev2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2.sliced(0, n * n / 2) = Devc.sliced(0, n * n / 2);  //  0.005292s
			hipDeviceSynchronize();
			std::cout << "| sliced     devc -> devc | " << Dev2.sliced(0, n * n / 2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2.strided(2) = Devc.strided(2);  //  0.005292s
			hipDeviceSynchronize();
			std::cout << "| strided    devc -> devc | " << Dev2.strided(2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2 = Host;
			hipDeviceSynchronize();
			std::cout << "| contiguous host -> host | " << Hos2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.sliced(0, n * n / 2) = Host.sliced(0, n * n / 2);  //  0.005292s
			hipDeviceSynchronize();
			std::cout << "| sliced     host -> host | " << Hos2.sliced(0, n * n / 2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.strided(2) = Host.strided(2);  //  0.005292s
			hipDeviceSynchronize();
			std::cout << "| strided    host -> host | " << Hos2.strided(2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
		}
		std::cout << "   " << std::endl;
	}

	std::cout << "line " << __LINE__ << std::endl;

	// BOOST_AUTO_TEST_CASE(thrust_copy_1D_issue123_complex)
	{
		using T = thrust::complex<double>;

		// static_assert(multi::is_trivially_default_constructible<T>{});
		// static_assert(std::is_trivially_copy_constructible_v<T>);
		// static_assert(std::is_trivially_assignable_v<T&, T>);

		multi::array<T, 1, test_allocator<T>> Devc(multi::extensions_t<1>{n * n});
		multi::array<T, 1, test_allocator<T>> Dev2(multi::extensions_t<1>{n * n});
		multi::array<T, 1>                    Host(multi::extensions_t<1>{n * n});
		std::iota(Host.elements().begin(), Host.elements().end(), 12.);
		multi::array<T, 1> Hos2(multi::extensions_t<1>{n * n});

		std::cout << "| 1D `" << typeid(T).name() << "` total data size: " << Host.num_elements() * sizeof(T) / 1073741824.0 << " GB | speed |\n|---|---|" << std::endl;
		{
			Devc = Host;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc = Host;
			hipDeviceSynchronize();
			std::cout << "| contiguous host -> devc | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc.sliced(0, n * n / 2) = Host.sliced(0, n * n / 2);
			hipDeviceSynchronize();
			std::cout << "| sliced     host -> devc | " << Host.sliced(0, n * n / 2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc.strided(2) = Host.strided(2);
			hipDeviceSynchronize();
			std::cout << "| strided    host -> devc | " << Host.strided(2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2 = Devc;
			hipDeviceSynchronize();
			std::cout << "| contiguous devc -> host | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST( Hos2 == Host );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.sliced(0, n * n / 2) = Devc.sliced(0, n * n / 2);
			hipDeviceSynchronize();
			std::cout << "| sliced     devc -> host | " << Host.sliced(0, n * n / 2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST( Hos2 == Host );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.strided(2) = Devc.strided(2);
			hipDeviceSynchronize();
			std::cout << "| strided    devc -> host | " << Host.strided(2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST( Hos2 == Host );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2 = Devc;
			hipDeviceSynchronize();
			std::cout << "| contiguous devc -> devc | " << Dev2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			auto                         Dev3 = Devc;
			hipDeviceSynchronize();
			std::cout << "| copy_ctr   devc -> devc | " << Devc.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev3 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			hipMemcpy(raw_pointer_cast(Dev2.data_elements()), raw_pointer_cast(Devc.data_elements()), Devc.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();
			std::cout << "| hipMemcpy devc -> devc | " << Dev2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			hipMemcpy(raw_pointer_cast(Dev2.data_elements()), raw_pointer_cast(Devc.data_elements()), Devc.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();
			std::cout << "| hipMemcpy devc -> devc | " << Dev2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2.sliced(0, n * n / 2) = Devc.sliced(0, n * n / 2);  //  0.005292s
			hipDeviceSynchronize();
			std::cout << "| sliced     devc -> devc | " << Dev2.sliced(0, n * n / 2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2.strided(2) = Devc.strided(2);  //  0.005292s
			hipDeviceSynchronize();
			std::cout << "| strided    devc -> devc | " << Dev2.strided(2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2 = Host;
			hipDeviceSynchronize();
			std::cout << "| contiguous host -> host | " << Hos2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.sliced(0, n * n / 2) = Host.sliced(0, n * n / 2);  //  0.005292s
			hipDeviceSynchronize();
			std::cout << "| sliced     host -> host | " << Hos2.sliced(0, n * n / 2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.strided(2) = Host.strided(2);  //  0.005292s
			hipDeviceSynchronize();
			std::cout << "| strided    host -> host | " << Hos2.strided(2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
		}
		std::cout << "   " << std::endl;
	}

	// BOOST_AUTO_TEST_CASE(thrust_cpugpu_2D_issue123_double)
	{
		using T = double;

		auto const exts = multi::extensions_t<2>({n, n});

		std::cout << "| 2D `" << typeid(T).name() << "` max data size " << exts.num_elements() * sizeof(T) / 1073741824.0 << " GB | speed |\n|---|---|" << std::endl;

		multi::array<T, 2, test_allocator<T>> Devc(exts);
		multi::array<T, 2, test_allocator<T>> Dev2(exts);

		multi::array<T, 2> Host(exts);
		std::iota(Host.elements().begin(), Host.elements().end(), 12.);
		multi::array<T, 2> Hos2(exts);

		{
			Devc({0, n/2}, {0, n/2}) = Host({0, n/2}, {0, n/2});  // 0.002859s
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc = Host;
			std::cout << "| contiguous host to devc | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc.sliced(0, n/2) = Host.sliced(0, n/2);  //  0.005292s
			std::cout << "| sliced     host to devc | " << Host.sliced(0, n/2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc({0, n/2}, {0, n/2}) = Host({0, n/2}, {0, n/2});  // 0.002859s
			std::cout << "| strided    host to devc | " << Host({0, n/2}, {0, n/2}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2 = Devc;
			std::cout << "| contiguous devc to host | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST(Hos2 == Host);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.sliced(0, n/2) = Devc.sliced(0, n/2);  //  0.005292s
			std::cout << "| sliced     devc to host | " << Host.sliced(0, n/2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST(Hos2 == Host);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2({0, n/2}, {0, n/2}) = Devc({0, n/2}, {0, n/2});  // 0.002859s
			std::cout << "| strided    devc to host | " << Host({0, n/2}, {0, n/2}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST(Hos2 == Host);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2 = Devc;
			hipDeviceSynchronize();
			std::cout << "| contiguous devc to devc | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			auto                         Dev3 = Devc;
			hipDeviceSynchronize();
			std::cout << "| copy_ctr   devc -> devc | " << Devc.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev3 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			hipMemcpy(raw_pointer_cast(Dev2.data_elements()), raw_pointer_cast(Devc.data_elements()), Devc.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();
			std::cout << "| hipMemcpy devc to devc | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			hipMemcpy(raw_pointer_cast(Dev2.data_elements()), raw_pointer_cast(Devc.data_elements()), Devc.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();
			std::cout << "| hipMemcpy devc to devc | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			auto                         Dev3 = Devc;
			hipDeviceSynchronize();
			std::cout << "| copy_ctr   devc to devc | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2.sliced(0, n/2) = Devc.sliced(0, n/2);  //  0.005292s
			hipDeviceSynchronize();
			std::cout << "| sliced     devc to devc | " << Host.sliced(0, n/2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2({0, n/2}, {0, n/2}) = Devc({0, n/2}, {0, n/2});  // 0.002859s
			hipDeviceSynchronize();
			std::cout << "| strided    devc to devc | " << Host({0, n/2}, {0, n/2}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2 = Host;
			std::cout << "| contiguous host to host | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.sliced(0, n/2) = Host.sliced(0, n/2);  //  0.005292s
			std::cout << "| sliced     host to host | " << Host.sliced(0, n/2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2({0, n/2}, {0, n/2}) = Host({0, n/2}, {0, n/2});  // 0.002859s
			std::cout << "| strided    host to host | " << Host({0, n/2}, {0, n/2}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		std::cout << "  " << std::endl;
	}

	// BOOST_AUTO_TEST_CASE(thrust_cpugpu_2D_issue123_complex)
	{
		using T = thrust::complex<double>;

		auto const exts = multi::extensions_t<2>({n, n});

		std::cout << "| 2D `" << typeid(T).name() << "` max data size " << exts.num_elements() * sizeof(T) / 1073741824.0 << " GB | speed |\n|---|---|" << std::endl;

		multi::array<T, 2, test_allocator<T>> Devc(exts);
		multi::array<T, 2, test_allocator<T>> Dev2(exts);

		multi::array<T, 2> Host(exts);
		std::iota(Host.elements().begin(), Host.elements().end(), 12.);
		multi::array<T, 2> Hos2(exts);

		{
			Devc({0, n/2}, {0, n/2}) = Host({0, n/2}, {0, n/2});  // 0.002859s
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc = Host;
			std::cout << "| contiguous host to devc | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc.sliced(0, n/2) = Host.sliced(0, n/2);  //  0.005292s
			std::cout << "| sliced     host to devc | " << Host.sliced(0, n/2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc({0, n/2}, {0, n/2}) = Host({0, n/2}, {0, n/2});  // 0.002859s
			std::cout << "| strided    host to devc | " << Host({0, n/2}, {0, n/2}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824.0 << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2 = Devc;
			std::cout << "| contiguous devc to host | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Hos2 == Host);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.sliced(0, n/2) = Devc.sliced(0, n/2);  //  0.005292s
			std::cout << "| sliced     devc to host | " << Host.sliced(0, n/2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Hos2 == Host);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2({0, n/2}, {0, n/2}) = Devc({0, n/2}, {0, n/2});  // 0.002859s
			std::cout << "| strided    devc to host | " << Host({0, n/2}, {0, n/2}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Hos2 == Host);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2 = Devc;
			hipDeviceSynchronize();
			std::cout << "| contiguous devc to devc | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			auto                         Dev3 = Devc;
			hipDeviceSynchronize();
			std::cout << "| copy_ctr   devc -> devc | " << Devc.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev3 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			hipMemcpy(raw_pointer_cast(Dev2.data_elements()), raw_pointer_cast(Devc.data_elements()), Devc.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();
			std::cout << "| hipMemcpy devc to devc | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			hipMemcpy(raw_pointer_cast(Dev2.data_elements()), raw_pointer_cast(Devc.data_elements()), Devc.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();
			std::cout << "| hipMemcpy devc to devc | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			auto                         Dev3 = Devc;
			hipDeviceSynchronize();
			std::cout << "| copy_ctr   devc to devc | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2.sliced(0, n/2) = Devc.sliced(0, n/2);  //  0.005292s
			hipDeviceSynchronize();
			std::cout << "| sliced     devc to devc | " << Host.sliced(0, n/2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2({0, n/2}, {0, n/2}) = Devc({0, n/2}, {0, n/2});  // 0.002859s
			hipDeviceSynchronize();
			std::cout << "| strided    devc to devc | " << Host({0, n/2}, {0, n/2}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2 = Host;
			std::cout << "| contiguous host to host | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.sliced(0, n/2) = Host.sliced(0, n/2);  //  0.005292s
			std::cout << "| sliced     host to host | " << Host.sliced(0, n/2).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2({0, n/2}, {0, n/2}) = Host({0, n/2}, {0, n/2});  // 0.002859s
			std::cout << "| strided    host to host | " << Host({0, n/2}, {0, n/2}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		std::cout << "  " << std::endl;
	}

	// BOOST_AUTO_TEST_CASE(thrust_cpugpu_issue123_3D_double)
	{
		using T         = double;
		auto const exts = multi::extensions_t<3>({1024, 1024, 100});

		std::cout << "| 3D `" << typeid(T).name() << "` max data size " << exts.num_elements() * sizeof(T) / 1073741824. << " GB | speed |\n|---|---|" << std::endl;

		multi::array<T, 3, test_allocator<T>> Devc(exts);
		multi::array<T, 3, test_allocator<T>> Dev2(exts);
		multi::array<T, 3>                    Host(exts);
		std::iota(Host.elements().begin(), Host.elements().end(), 12.);
		multi::array<T, 3> Hos2(exts);

		{
			Devc({0, 512}, {0, 512}, {0, 512}) = Host({0, 512}, {0, 512}, {0, 512});  // 0.002859s
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc = Host;
			std::cout << "| contiguous host to devc | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << " GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc.sliced(0, 512) = Host.sliced(0, 512);  //  0.005292s
			std::cout << "| sliced     host to devc | " << Host.sliced(0, 512).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << " GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc({0, 512}, {0, 512}, {0, 512}) = Host({0, 512}, {0, 512}, {0, 512});  // 0.002859s
			std::cout << "| strided    host to devc | " << Host({0, 512}, {0, 512}, {0, 512}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2 = Devc;
			std::cout << "| contiguous devc to host | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Hos2 == Host);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.sliced(0, 512) = Devc.sliced(0, 512);  //  0.005292s
			std::cout << "| sliced     devc to host | " << Host.sliced(0, 512).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Hos2 == Host);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2({0, 512}, {0, 512}, {0, 512}) = Devc({0, 512}, {0, 512}, {0, 512});  // 0.002859s
			std::cout << "| strided    devc to host | " << Host({0, 512}, {0, 512}, {0, 512}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Hos2 == Host);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2 = Devc;
			hipDeviceSynchronize();
			std::cout << "| contiguous devc to devc | " << Dev2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Dev2 == Devc);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			auto                         Dev3 = Devc;
			hipDeviceSynchronize();
			std::cout << "| copy_ctr   devc -> devc | " << Devc.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev3 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			hipMemcpy(raw_pointer_cast(Dev2.data_elements()), raw_pointer_cast(Devc.data_elements()), Devc.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();
			std::cout << "| hipMemcpy devc -> devc | " << Dev2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			hipMemcpy(raw_pointer_cast(Dev2.data_elements()), raw_pointer_cast(Devc.data_elements()), Devc.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();
			std::cout << "| hipMemcpy devc -> devc | " << Dev2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2.sliced(0, 512) = Devc.sliced(0, 512);  //  0.005292s
			hipDeviceSynchronize();
			std::cout << "| sliced     devc to devc | " << Dev2.sliced(0, 512).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Dev2 == Devc);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2({0, 512}, {0, 512}, {0, 512}) = Devc({0, 512}, {0, 512}, {0, 512});  // 0.002859s
			hipDeviceSynchronize();
			std::cout << "| strided    devc to devc | " << Dev2({0, 512}, {0, 512}, {0, 512}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Dev2 == Devc);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2 = Host;
			std::cout << "| contiguous host to host | " << Hos2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.sliced(0, 512) = Host.sliced(0, 512);  //  0.005292s
			std::cout << "| sliced     host to host | " << Hos2.sliced(0, 512).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2({0, 512}, {0, 512}, {0, 512}) = Host({0, 512}, {0, 512}, {0, 512});  // 0.002859s
			std::cout << "| strided    host to host | " << Hos2({0, 512}, {0, 512}, {0, 512}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		std::cout << "   " << std::endl;
	}

	// BOOST_AUTO_TEST_CASE(thrust_cpugpu_issue123_3D_complex)
	{
		using T         = thrust::complex<double>;
		auto const exts = multi::extensions_t<3>({1024, 1024, 100});

		std::cout << "| 3D `" << typeid(T).name() << "` max data size " << exts.num_elements() * sizeof(T) / 1073741824. << " GB | speed |\n|---|---|" << std::endl;

		multi::array<T, 3, test_allocator<T>> Devc(exts);
		multi::array<T, 3, test_allocator<T>> Dev2(exts);
		multi::array<T, 3>                    Host(exts);
		std::iota(Host.elements().begin(), Host.elements().end(), 12.);
		multi::array<T, 3> Hos2(exts);

		{
			Devc({0, 512}, {0, 512}, {0, 512}) = Host({0, 512}, {0, 512}, {0, 512});  // 0.002859s
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc = Host;
			std::cout << "| contiguous host to devc | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << " GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc.sliced(0, 512) = Host.sliced(0, 512);  //  0.005292s
			std::cout << "| sliced     host to devc | " << Host.sliced(0, 512).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << " GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Devc({0, 512}, {0, 512}, {0, 512}) = Host({0, 512}, {0, 512}, {0, 512});  // 0.002859s
			std::cout << "| strided    host to devc | " << Host({0, 512}, {0, 512}, {0, 512}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2 = Devc;
			std::cout << "| contiguous devc to host | " << Host.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Hos2 == Host);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.sliced(0, 512) = Devc.sliced(0, 512);  //  0.005292s
			std::cout << "| sliced     devc to host | " << Host.sliced(0, 512).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Hos2 == Host);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2({0, 512}, {0, 512}, {0, 512}) = Devc({0, 512}, {0, 512}, {0, 512});  // 0.002859s
			std::cout << "| strided    devc to host | " << Host({0, 512}, {0, 512}, {0, 512}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Hos2 == Host);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2 = Devc;
			hipDeviceSynchronize();
			std::cout << "| contiguous devc to devc | " << Dev2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Dev2 == Devc);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			auto                         Dev3 = Devc;
			hipDeviceSynchronize();
			std::cout << "| copy_ctr   devc -> devc | " << Devc.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev3 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			hipMemcpy(raw_pointer_cast(Dev2.data_elements()), raw_pointer_cast(Devc.data_elements()), Devc.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();
			std::cout << "| hipMemcpy devc -> devc | " << Dev2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			hipMemcpy(raw_pointer_cast(Dev2.data_elements()), raw_pointer_cast(Devc.data_elements()), Devc.num_elements() * sizeof(T), hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();
			std::cout << "| hipMemcpy devc -> devc | " << Dev2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST( Dev2 == Devc );
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2.sliced(0, 512) = Devc.sliced(0, 512);  //  0.005292s
			hipDeviceSynchronize();
			std::cout << "| sliced     devc to devc | " << Dev2.sliced(0, 512).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Dev2 == Devc);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Dev2({0, 512}, {0, 512}, {0, 512}) = Devc({0, 512}, {0, 512}, {0, 512});  // 0.002859s
			hipDeviceSynchronize();
			std::cout << "| strided    devc to devc | " << Dev2({0, 512}, {0, 512}, {0, 512}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
			// BOOST_TEST(Dev2 == Devc);
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2 = Host;
			std::cout << "| contiguous host to host | " << Hos2.num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2.sliced(0, 512) = Host.sliced(0, 512);  //  0.005292s
			std::cout << "| sliced     host to host | " << Hos2.sliced(0, 512).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		{
			boost::timer::auto_cpu_timer t{""};
			Hos2({0, 512}, {0, 512}, {0, 512}) = Host({0, 512}, {0, 512}, {0, 512});  // 0.002859s
			std::cout << "| strided    host to host | " << Hos2({0, 512}, {0, 512}, {0, 512}).num_elements() * sizeof(T) / (t.elapsed().wall / 1e9) / 1073741824. << "GB/sec |" << std::endl;
		}
		std::cout << "   " << std::endl;
	}
#endif

	std::cout << "line " << __LINE__ << std::endl;

	return boost::report_errors();
}
