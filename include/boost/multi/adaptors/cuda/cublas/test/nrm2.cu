#include "hip/hip_runtime.h"
// Copyright 2024 Alfredo A. Correa
// Distributed under the Boost Software License, Version 1.0.
// https://www.boost.org/LICENSE_1_0.txt

#include <boost/multi/adaptors/cuda/cublas.hpp>

#include <boost/multi/adaptors/blas/dot.hpp>
#include <boost/multi/adaptors/blas/axpy.hpp>
#include <boost/multi/adaptors/blas/gemm.hpp>
#include <boost/multi/adaptors/blas/nrm2.hpp>
#include <boost/multi/adaptors/blas/scal.hpp>

#include <boost/multi/adaptors/thrust.hpp>

#include<thrust/complex.h>

#include<numeric>

namespace multi = boost::multi;

#include <boost/core/lightweight_test.hpp>
#define BOOST_AUTO_TEST_CASE(CasenamE) /**/

int main() {
// BOOST_AUTO_TEST_CASE(cublas_dot_out_param_complex_C) {
//  namespace blas = multi::blas;
//  using complex = thrust::complex<double>;
//  complex const I{0.0, 1.0};

//  multi::thrust::cuda::array<complex, 1> const x = {1.0 + 0.0*I, 2.0 + 0.0*I, 3.0 + 0.0*I};  // NOLINT(readability-identifier-length) BLAS naming
//  multi::thrust::cuda::array<complex, 1> const y = {1.0 + 0.0*I, 2.0 + 2.0*I, 3.0 + 0.0*I};  // NOLINT(readability-identifier-length) BLAS naming

//  complex res{0.0, 0.0};
//  blas::dot(blas::C(x), y, res);
// //  BOOST_TEST( res == std::inner_product(begin(x), end(x), begin(y), complex{0.0, 0.0}, std::plus<>{}, [](auto const& alpha, auto const& omega) {return conj(alpha)*omega;}) );
// }

BOOST_AUTO_TEST_CASE(cublas_dot_out_array0D_complex_C) {
	namespace blas = multi::blas;
	using complex = thrust::complex<double>;
	complex const I{0.0, 1.0};

	multi::thrust::cuda::array<complex, 1> const x = {1.0 + 0.0*I, 2.0 + 0.0*I, 3.0 + 0.0*I};  // NOLINT(readability-identifier-length) BLAS naming
	multi::thrust::cuda::array<complex, 1> const y = {1.0 + 0.0*I, 2.0 + 2.0*I, 3.0 + 0.0*I};  // NOLINT(readability-identifier-length) BLAS naming

	multi::thrust::cuda::array<complex, 0> res{complex{0.0, 0.0}};
	blas::dot(blas::C(x), y, res);

	{
		multi::array<complex, 0> res_copy{complex{0.0, 0.0}};
		res_copy = res;
		BOOST_TEST(( *res_copy.base() == complex{14.0, 4.0} ));
	}
	{
		multi::array<complex, 0> res_copy{res};
		BOOST_TEST(( *res_copy.base() == complex{14.0, 4.0} ));
	}
}

BOOST_AUTO_TEST_CASE(cublas_dot_out_array0D_complex_C) {
	namespace blas = multi::blas;
	using complex = thrust::complex<double>;
	complex const I{0.0, 1.0};

	multi::thrust::cuda::array<complex, 1> const x = {1.0 + 0.0*I, 2.0 + 0.0*I, 3.0 + 0.0*I};  // NOLINT(readability-identifier-length) BLAS naming
	multi::thrust::cuda::array<complex, 1> const y = {1.0 + 0.0*I, 2.0 + 2.0*I, 3.0 + 0.0*I};  // NOLINT(readability-identifier-length) BLAS naming

	multi::thrust::cuda::array<complex, 0> res{complex{0.0, 0.0}};

	{
		double res{0.0};
		multi::array<double, 1> const xx = {1.0, 2.0, 3.0};

		res = blas::nrm2(xx);
		BOOST_TEST( std::abs( res - std::sqrt(1.0*1.0 + 2.0*2.0 + 3.0*3.0) ) < 1e-7 );
	}
	{
		double res{0.0};
		multi::array<double, 1> const xx = {1.0, 2.0, 3.0};

		blas::nrm2(xx, res);
		BOOST_TEST( std::abs( res - std::sqrt(1.0*1.0 + 2.0*2.0 + 3.0*3.0) ) < 1e-7 );
	}
	{
		multi::array<double, 0> res{0.0};
		multi::array<double, 1> const xx = {1.0, 2.0, 3.0};

		blas::nrm2(xx, res);
		BOOST_TEST( std::abs( *res.base() - std::sqrt(1.0*1.0 + 2.0*2.0 + 3.0*3.0) ) < 1e-7 );
	}
	{
		multi::array<double, 0> res{0.0};
		multi::array<std::complex<double>, 1> const xx = {std::complex<double>{1.0, 2.0}, std::complex<double>{3.0, 4.0}, std::complex<double>{5.0, 6.0}};

		blas::nrm2(xx, res);
		BOOST_TEST( std::abs( *res.base() - std::sqrt(std::norm(std::complex<double>{1.0, 2.0}) + std::norm(std::complex<double>{3.0, 4.0}) + std::norm(std::complex<double>{5.0, 6.0})) ) < 1e-7 );
	}
	{
		multi::thrust::cuda::array<double, 0> res{0.0};
		multi::thrust::cuda::array<double, 1> const xx = {1.0, 2.0, 3.0};

		blas::nrm2(xx, res);
		BOOST_TEST( *res.base() == std::sqrt(1.0*1.0 + 2.0*2.0 + 3.0*3.0) );
	}
	{
		multi::thrust::cuda::array<double, 0> res{0.0};
		multi::thrust::cuda::array<double, 1> const xx = {1.0, 2.0, 3.0};

		res = blas::nrm2(xx);
		BOOST_TEST( *res.base() == std::sqrt(1.0*1.0 + 2.0*2.0 + 3.0*3.0) );

		double res2 = blas::nrm2(xx);
		BOOST_TEST( res2 == std::sqrt(1.0*1.0 + 2.0*2.0 + 3.0*3.0) );
	}
	{
		multi::thrust::cuda::array<double, 0> res{0.0};
		multi::thrust::cuda::array<thrust::complex<double>, 1> const xx = {thrust::complex<double>(1.0, 2.0), thrust::complex<double>(3.0, 4.0), thrust::complex<double>(4.0, 5.0)};

		res = blas::nrm2(xx);
		BOOST_TEST( *res.base() == std::sqrt(std::norm(std::complex<double>(1.0, 2.0)) + std::norm(std::complex<double>(3.0, 4.0)) + std::norm(std::complex<double>(4.0, 5.0))) );

		double res2 = blas::nrm2(xx);
		BOOST_TEST( res2 == *res.base() );

		auto res3 = blas::nrm2(xx);
	}
	{
		multi::thrust::cuda::array<double, 0> res{0.0};
		multi::thrust::cuda::array<thrust::complex<double>, 1> const xx = {thrust::complex<double>(1.0, 2.0), thrust::complex<double>(3.0, 4.0), thrust::complex<double>(4.0, 5.0)};

		blas::nrm2_n(xx.begin(), xx.size(), res.base());
		BOOST_TEST( *res.base() == std::sqrt(std::norm(std::complex<double>(1.0, 2.0)) + std::norm(std::complex<double>(3.0, 4.0)) + std::norm(std::complex<double>(4.0, 5.0))) );
	}
	{
		multi::thrust::cuda::array<double, 0> res{0.0};
		multi::thrust::cuda::array<double, 1> const xx = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0};

		blas::nrm2_n(xx.begin(), 3, res.base());
		BOOST_TEST( *res.base() == std::sqrt(1.0*1.0 + 2.0*2.0 + 3.0*3.0) );
	}

	// {
	//  multi::thrust::cuda::array<double, 0> res{0.0};
	//  multi::thrust::cuda::array<double, 1> const xx = {1.0, 2.0, 3.0};

	//  blas::nrm2(xx, res);
	//  BOOST_TEST( *res.base() == std::sqrt(1.0*1.0 + 2.0*2.0 + 3.0*3.0) );
	// }

	// multi::thrust::cuda::array<complex, 0> res{complex{0.0, 0.0}};
	// blas::dot(blas::C(x), y, res);

	// {
	//  multi::array<complex, 0> res_copy{complex{0.0, 0.0}};
	//  res_copy = res;
	//  BOOST_TEST(( *res_copy.base() == complex{14.0, 4.0} ));
	// }
	// {
	//  multi::array<complex, 0> res_copy{res};
	//  BOOST_TEST(( *res_copy.base() == complex{14.0, 4.0} ));
	// }
}

// BOOST_AUTO_TEST_CASE(blas_dot_functional_complex_C) {
//  namespace blas = multi::blas;
//  using complex = thrust::complex<double>;
//  complex const I{0.0, 1.0};

//  multi::thrust::cuda::array<complex, 1> const x = {1.0 + 0.0*I, 2.0 + 0.0*I, 3.0 + 0.0*I};  // NOLINT(readability-identifier-length) BLAS naming
//  multi::thrust::cuda::array<complex, 1> const y = {1.0 + 0.0*I, 2.0 + 2.0*I, 3.0 + 0.0*I};  // NOLINT(readability-identifier-length) BLAS naming

//  complex res = blas::dot(blas::C(x), y);
//  BOOST_TEST( res == std::inner_product(begin(x), end(x), begin(y), complex{0.0, 0.0}, std::plus<>{}, [](auto const& alpha, auto const& omega) {return conj(alpha)*omega;}) );
// }

// BOOST_AUTO_TEST_CASE(blas_dot_functional_mutate_complex_C) {
//  namespace blas = multi::blas;
//  using complex = thrust::complex<double>;
//  complex const I{0.0, 1.0};

//  multi::thrust::cuda::array<complex, 1> const x = {1.0 + 0.0*I, 2.0 + 0.0*I, 3.0 + 0.0*I};  // NOLINT(readability-identifier-length) BLAS naming
//  multi::thrust::cuda::array<complex, 1> const y = {1.0 + 0.0*I, 2.0 + 2.0*I, 3.0 + 0.0*I};  // NOLINT(readability-identifier-length) BLAS naming

//  complex res;
//  res = blas::dot(blas::C(x), y);
//  BOOST_TEST( res == std::inner_product(begin(x), end(x), begin(y), complex{0.0, 0.0}, std::plus<>{}, [](auto const& alpha, auto const& omega) {return conj(alpha)*omega;}) );
// }

return boost::report_errors();}
