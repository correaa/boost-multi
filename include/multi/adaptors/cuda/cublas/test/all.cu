#include "hip/hip_runtime.h"
// -*-indent-tabs-mode:t;c-basic-offset:4;tab-width:4;autowrap:nil;-*-
// Copyright 2023 Alfredo A. Correa

#define BOOST_TEST_MODULE "C++ Unit Tests for Multi CUBLAS all"
#include<boost/test/unit_test.hpp>

#include <multi/adaptors/cuda/cublas.hpp>

#include <multi/adaptors/blas/asum.hpp>
#include <multi/adaptors/blas/axpy.hpp>
#include <multi/adaptors/blas/copy.hpp>
#include <multi/adaptors/blas/gemm.hpp>
#include <multi/adaptors/blas/nrm2.hpp>
#include <multi/adaptors/blas/scal.hpp>
#include <multi/adaptors/blas/swap.hpp>

#include <multi/adaptors/thrust.hpp>

#include<thrust/complex.h>

#include<numeric>
#include <thrust/inner_product.h>
#include <thrust/transform_reduce.h>

namespace multi = boost::multi;

using complex = thrust::complex<double>;

template<class T = complex, class Alloc = std::allocator<T>>
auto generate_ABx() {
	complex const I{0.0, 1.0};
	multi::array<T, 1, Alloc> x = { 1.0 + I*0.0,  2.0 + I*0.0,  3.0 + I*0.0,  4.0 + I*0.0};

	multi::array<complex, 2, Alloc> A = {
		{ 1.0 + I*0.0,  2.0 + I*0.0,  3.0 + I*0.0,  4.0 + I*0.0},
		{ 5.0 + I*0.0,  6.0 + I*0.0,  7.0 + I*0.0,  8.0 + I*0.0},
		{ 9.0 + I*0.0, 10.0 + I*0.0, 11.0 + I*0.0, 12.0 + I*0.0},
		{13.0 + I*0.0, 14.0 + I*0.0, 15.0 + I*0.0, 16.0 + I*0.0},
	};

	multi::array<complex, 2, Alloc> B = {
		{ 1.0 + I*0.0,  2.0 + I*0.0,  3.0 + I*0.0,  4.0 + I*0.0},
		{ 5.0 + I*0.0,  6.0 + I*0.0,  7.0 + I*0.0,  8.0 + I*0.0},
		{ 9.0 + I*0.0, 10.0 + I*0.0, 11.0 + I*0.0, 12.0 + I*0.0},
		{13.0 + I*0.0, 14.0 + I*0.0, 15.0 + I*0.0, 16.0 + I*0.0},
	};

	return std::make_tuple(std::move(x), std::move(A), std::move(B));
}

BOOST_AUTO_TEST_CASE(cublas_scal_complex_column) {
	namespace blas = multi::blas;
	complex const I{0.0, 1.0};

	{
		using T = complex;
		auto [x, A, B] = generate_ABx<T, thrust::cuda::allocator<T> >();
		auto const s = 2.0 + I*3.0;
		blas::scal(s, x);  // x_i <- s*x_i

		{
			auto [x2, A2, B2] = generate_ABx<complex, thrust::cuda::allocator<complex> >();
			auto xx = +x2;
			blas::scal(s, xx);
			BOOST_REQUIRE(xx == x);
		}
		{
			auto [x2, A2, B2] = generate_ABx<complex, thrust::cuda::allocator<complex> >();
			using blas::operators::operator*=;
			x2 *= s;
			BOOST_REQUIRE(x == x2);
		}
		{
			auto [x2, A2, B2] = generate_ABx<complex, thrust::cuda::allocator<complex> >();
			thrust::transform(x2.begin(), x2.end(), x2.begin(), [s] __device__ (T& e) {return s*e;});

			BOOST_REQUIRE(x == x2);
		}
		{
			auto [x2, A2, B2] = generate_ABx<complex, thrust::cuda::allocator<complex> >();
			thrust::for_each(x2.begin(), x2.end(), [s] __device__ (T& e) {return e*=s;});

			BOOST_REQUIRE(x == x2);
		}
	}
}

BOOST_AUTO_TEST_CASE(cublas_copy_complex) {
	namespace blas = multi::blas;
	complex const I{0.0, 1.0};

	using T = complex;
	using Alloc = thrust::cuda::allocator<complex>;

	multi::array<T, 1, Alloc> const x = { 1.0 + I*8.0,  2.0 + I*6.0,  3.0 + I*5.0,  4.0 + I*3.0};
	multi::array<T, 1, Alloc> y = { 1.0 + I*9.0,  2.0 + I*6.0,  3.0 + I*5.0,  4.0 + I*3.0};

	blas::copy(x, y);
	BOOST_REQUIRE( static_cast<complex>(y[0]) == 1.0 + I*8.0 );
	{
		thrust::copy(begin(x), end(x), begin(y));
		BOOST_REQUIRE( static_cast<complex>(y[0]) == 1.0 + I*8.0 );
	}
	{
		blas::copy_n(x.begin(), x.size(), y.begin());
		BOOST_REQUIRE( static_cast<complex>(y[0]) == 1.0 + I*8.0 );
	}
	{
		y() = blas::copy(x);
		BOOST_REQUIRE( static_cast<complex>(y[0]) == 1.0 + I*8.0 );
	}
	{
		multi::array<T, 1, Alloc> yy = blas::copy(x);
		BOOST_REQUIRE( static_cast<complex>(yy[0]) == 1.0 + I*8.0 );
	}
	{
		y = blas::copy(x);
		BOOST_REQUIRE( static_cast<complex>(y[0]) == 1.0 + I*8.0 );
	}
	{
		{
			using blas::operators::operator<<;
			y << x;
		//  BOOST_REQUIRE(( static_cast<complex>(y[0]) == 1.0 + I*8.0 ));  // this can't be used with a free operator<<
		}
		BOOST_REQUIRE(( static_cast<complex>(y[0]) == 1.0 + I*8.0 ));  // this can't be used with a free operator<<
	}
}

#if 1
BOOST_AUTO_TEST_CASE(cublas_swap_complex) {
	namespace blas = multi::blas;
	complex const I{0.0, 1.0};

	using T = complex;
	using Alloc = thrust::cuda::allocator<complex>;

	multi::array<T, 1, Alloc> x = { 1.0 + I*8.0,  2.0 + I*6.0,  3.0 + I*5.0,  4.0 + I*3.0};
	multi::array<T, 1, Alloc> y = { 1.0 + I*9.0,  2.0 + I*6.0,  3.0 + I*5.0,  4.0 + I*3.0};

	blas::swap(x, y);
	BOOST_REQUIRE( static_cast<complex>(x[0]) == 1.0 + I*9.0 );
	{
		thrust::swap_ranges(begin(x), end(x), begin(y));
		thrust::swap_ranges(begin(x), end(x), begin(y));
		BOOST_REQUIRE( static_cast<complex>(x[0]) == 1.0 + I*9.0 );
	}
	{
		using blas::operator^;
		(x^y);
		(x^y);
		BOOST_REQUIRE( static_cast<complex>(x[0]) == 1.0 + I*9.0 );
	}
}

BOOST_AUTO_TEST_CASE(cublas_asum_complex_column) {
	namespace blas = multi::blas;
	complex const I{0.0, 1.0};

	using T = complex;
	using Alloc = thrust::cuda::allocator<complex>;

	multi::array<T, 1, Alloc> const x = { 1.0 + I*8.0,  2.0 + I*6.0,  3.0 + I*5.0,  4.0 + I*3.0};

	double res;
	blas::asum_n(x.begin(), x.size(), &res);
	{
		double res2;
		res2 = blas::asum(x);
		BOOST_REQUIRE( res == res2 );
	}
	{
		double res2 = blas::asum(x);
		BOOST_REQUIRE( res == res2 );
	}
	{
		auto res2 = std::transform_reduce(
			x.begin(), x.end(), double{}, std::plus<>{}, [](T const& e) {return std::abs(e.real()) + std::abs(e.imag());}
		);
		BOOST_REQUIRE( res == res2 );
	}
	{
		auto res2 = thrust::transform_reduce(
			x.begin(), x.end(), [] __device__ (T const& e) {return std::abs(e.real()) + std::abs(e.imag());},
			double{}, thrust::plus<>{}
		);
		BOOST_REQUIRE( res == res2 );
	}
	{
		multi::static_array<double, 0, thrust::cuda::allocator<double>> res2({}, 0.0);
		res2.assign( &blas::asum(x) );
		res2 = blas::asum(x);
		BOOST_REQUIRE(( res == static_cast<multi::static_array<double, 0, thrust::cuda::allocator<double>>::element_ref>(res2) ));
		BOOST_REQUIRE(( res == static_cast<double>(res2) ));
	//  BOOST_REQUIRE( res == res2 );
	}
	{
		multi::array<double, 0, thrust::cuda::allocator<double>> res2 = blas::asum(x);
		BOOST_REQUIRE(( res == static_cast<multi::static_array<double, 0, thrust::cuda::allocator<double>>::element_ref>(res2) ));
		BOOST_REQUIRE(( res == static_cast<double>(res2) ));
	//  BOOST_REQUIRE( res == res2 );
	}
	{
		using blas::operators::operator==;
		using blas::operators::operator!=;
		BOOST_REQUIRE( x != 0 );
		BOOST_REQUIRE( not (x == 0) );
	}
	{
		using blas::operators::contains_nan;
		BOOST_REQUIRE( not contains_nan(x) );
	}
	{
		using blas::operators::isfinite;
		using blas::operators::isinf;
		BOOST_REQUIRE( isfinite(x) );
		BOOST_REQUIRE( not isinf(x) );
	}
}

BOOST_AUTO_TEST_CASE(cublas_asum_complex_nans) {
	namespace blas = multi::blas;
	complex const I{0.0, 1.0};

	using T = complex;
	using Alloc = thrust::cuda::allocator<complex>;

	multi::array<T, 1, Alloc> const x = { 1.0 + I*8.0,  std::numeric_limits<double>::quiet_NaN() + I*6.0,  3.0 + I*5.0,  4.0 + I*3.0};

	double res;
	{
		using blas::operators::contains_nan;
		BOOST_REQUIRE( contains_nan(x) );
	}
	{
		using blas::operators::operator==;
		using blas::operators::operator!=;
		BOOST_REQUIRE( not (x != 0) );
		BOOST_REQUIRE( not (x == 0) );
	}
	{
		using blas::operators::isfinite;
		using blas::operators::isinf;
		BOOST_REQUIRE( not isfinite(x) );
		BOOST_REQUIRE( not isinf(x) );
	}
}

BOOST_AUTO_TEST_CASE(cublas_asum_complex_inf) {
	namespace blas = multi::blas;
	complex const I{0.0, 1.0};

	using T = complex;
	using Alloc = thrust::cuda::allocator<complex>;

	multi::array<T, 1, Alloc> const x = { 1.0 + I*8.0,  std::numeric_limits<double>::infinity() + I*6.0,  3.0 + I*5.0,  4.0 + I*3.0};

	double res;
	{
		using blas::operators::contains_nan;
		BOOST_REQUIRE( not contains_nan(x) );
	}
	{
		using blas::operators::operator==;
		using blas::operators::operator!=;
		BOOST_REQUIRE(     (x != 0) );
		BOOST_REQUIRE( not (x == 0) );
	}
	{
		using blas::operators::isfinite;
		using blas::operators::isinf;
		BOOST_REQUIRE( not isfinite(x) );
		BOOST_REQUIRE( isinf(x) );
	}
}

BOOST_AUTO_TEST_CASE(cublas_nrm2_complex_column) {
	namespace blas = multi::blas;
	complex const I{0.0, 1.0};

	using T = complex;
	using Alloc =  thrust::cuda::allocator<complex>;

	multi::array<T, 1, Alloc> const x = { 1.0 + I*8.0,  2.0 + I*6.0,  3.0 + I*5.0,  4.0 + I*3.0};

	double res;
	blas::nrm2(x, res);
	{
		double res2;
		res2 = blas::nrm2(x);
		BOOST_REQUIRE( res == res2 );
	}
	{
		auto res2 = +blas::nrm2(x);
		BOOST_REQUIRE( res == res2 );
	}
	{
		auto res2 = sqrt(thrust::transform_reduce(
			x.begin(), x.end(), [] __device__ (T const& e) {return thrust::norm(e);},
			double{}, thrust::plus<>{}
		));
		BOOST_REQUIRE( res == res2 );
	}
	{
		multi::array<double, 0, thrust::cuda::allocator<double>> res2 = blas::nrm2(x);
		BOOST_REQUIRE(( res == static_cast<double>(res2) ));
	}
}

BOOST_AUTO_TEST_CASE(cublas_dot_complex_column) {
	namespace blas = multi::blas;
	complex const I{0.0, 1.0};

	using T = complex;
	using Alloc =  thrust::cuda::allocator<complex>;

	multi::array<T, 1, Alloc> const x = { 1.0 + I*8.0,  2.0 + I*6.0,  3.0 + I*5.0,  4.0 + I*3.0};
	multi::array<T, 1, Alloc> const y = { 1.0 + I*2.0,  2.0 + I*3.0,  3.0 + I*5.0,  4.0 + I*7.0};

	{
		T res;
		blas::dot(x, y, res);
		{
			complex res2;
			res2 = blas::dot(x, y);
			BOOST_REQUIRE(res == res2);
		}
		{
			multi::array<complex, 0> res2(complex{1.0, 0.0});
			res2 = blas::dot(x, y);
			BOOST_REQUIRE( static_cast<complex>(res2) == res );
		}
		{
			using blas::operators::operator,;
			auto res2 = +(x, y);
			BOOST_REQUIRE(res == res2);
		}
		{
			auto res2 = +blas::dot(x, y);
			BOOST_REQUIRE(res == res2);
		}
		{
		//  auto [x2, A2, B2] = generate_ABx<complex, thrust::cuda::allocator<complex> >();
		//  thrust::for_each(x2.begin(), x2.end(), [s] __device__ (T& e) {return e*=s;});
			auto res2 = thrust::inner_product(x.begin(), x.end(), y.begin(), T{});
			BOOST_REQUIRE(res == res2);
		}
	}
	{
		T res;
		blas::dot(blas::C(x), y, res);
		{
			using blas::operators::operator,;
			using blas::operators::operator*;
			auto res2 = +(*x, y);
			BOOST_REQUIRE(res == res2);
		}
		{
			auto res2 = +blas::dot(blas::C(x), y);
			BOOST_REQUIRE(res == res2);
		}
		{
		//  auto [x2, A2, B2] = generate_ABx<complex, thrust::cuda::allocator<complex> >();
		//  thrust::for_each(x2.begin(), x2.end(), [s] __device__ (T& e) {return e*=s;});
			auto res2 = thrust::inner_product(x.begin(), x.end(), y.begin(), T{}, thrust::plus<>{}, [] __device__ (T const& t1, T const& t2) {return conj(t1)*t2;});
			BOOST_REQUIRE(res == res2);
		}
	}
	{
		T res;
		blas::dot(x, blas::C(y), res);
		{
			using blas::operators::operator,;
			auto res2 = +(x, blas::C(y));
			BOOST_REQUIRE(res == res2);
		}
		{
			auto res2 = +blas::dot(x, blas::C(y));
			BOOST_REQUIRE(res == res2);
		}
		{
		//  auto [x2, A2, B2] = generate_ABx<complex, thrust::cuda::allocator<complex> >();
		//  thrust::for_each(x2.begin(), x2.end(), [s] __device__ (T& e) {return e*=s;});
			auto res2 = thrust::inner_product(x.begin(), x.end(), y.begin(), T{}, thrust::plus<>{}, [] __device__ (T const& t1, T const& t2) {return t1*conj(t2);});
			BOOST_REQUIRE(res == res2);
		}
		{
			BOOST_REQUIRE( blas::dot(blas::C(x), x) == pow(blas::nrm2(x), 2.0) );
			BOOST_REQUIRE( blas::dot(x, blas::C(x)) == pow(blas::nrm2(x), 2.0) );

			using blas::operators::operator,;
			using blas::operators::operator*;
			using blas::operators::abs;
			using blas::operators::norm;
			using blas::operators::operator^;

			BOOST_REQUIRE( (*x, x) == pow(abs(x), 2.0) );
			BOOST_REQUIRE( (*x, x) == pow(abs(x), 2)   );
			BOOST_REQUIRE( (*x, x) == norm(x)          );

			BOOST_REQUIRE( (x, *x) == pow(abs(x), 2.0) );
			BOOST_REQUIRE( (x, *x) == pow(abs(x), 2)   );
			BOOST_REQUIRE( (x, *x) == norm(x)          );

			BOOST_REQUIRE( (*x, x) == (x^2)            );
		}
	}
}

BOOST_AUTO_TEST_CASE(cublas_axpy_complex_one) {
	namespace blas = multi::blas;
	complex const I{0.0, 1.0};

	using T = complex;
	using Alloc =  thrust::cuda::allocator<complex>;

	multi::array<complex, 1, Alloc> const x = { {1.1, 0.0}, {2.1, 0.0}, {3.1, 0.0}, {4.1, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 1, Alloc> y = { {2.1, 0.0}, {4.1, 0.0}, {6.1, 0.0}, {11.0, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming

	blas::axpy(1.0, x, y);
	std::cout << y[0] << std::endl;
	BOOST_REQUIRE( static_cast<complex>(y[0]) == 3.2 + I*0.0 );
	{
		multi::array<complex, 1, Alloc> yy = { {2.1, 0.0}, {4.1, 0.0}, {6.1, 0.0}, {11.0, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
		thrust::transform(x.begin(), x.end(), yy.begin(), yy.begin(), [] __device__ (auto const& ex, auto const& ey) {return ex + ey;});
		BOOST_TEST( yy == y , boost::test_tools::per_element() );
	}
	{
		multi::array<complex, 1, Alloc> yy = { {2.1, 0.0}, {4.1, 0.0}, {6.1, 0.0}, {11.0, 0.0} };
		using blas::operators::operator+=;
		yy += x;
		BOOST_REQUIRE( yy == y );
	}
}

BOOST_AUTO_TEST_CASE(cublas_axpy_complex_mone) {
	namespace blas = multi::blas;
	complex const I{0.0, 1.0};

	using T = complex;
	using Alloc =  thrust::cuda::allocator<complex>;

	multi::array<complex, 1, Alloc> const x = { {1.1, 0.0}, {2.1, 0.0}, {3.1, 0.0}, {4.1, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 1, Alloc> y = { {2.1, 0.0}, {4.1, 0.0}, {6.1, 0.0}, {11.0, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming

	blas::axpy(-1.0, x, y);
	std::cout << y[0] << std::endl;
	BOOST_REQUIRE( static_cast<complex>(y[0]) == 1.0 + I*0.0 );
	{
		multi::array<complex, 1, Alloc> yy = { {2.1, 0.0}, {4.1, 0.0}, {6.1, 0.0}, {11.0, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
		thrust::transform(x.begin(), x.end(), yy.begin(), yy.begin(), [] __host__ __device__ (T ex, T ey) {return -1.0*ex + ey;});
		BOOST_TEST( yy == y , boost::test_tools::per_element() );
	}
	{
		multi::array<complex, 1, Alloc> yy = { {2.1, 0.0}, {4.1, 0.0}, {6.1, 0.0}, {11.0, 0.0} };
		using blas::operators::operator-=;
		yy -= x;
		BOOST_REQUIRE( yy == y );
	}
	{
		multi::array<complex, 1, Alloc> yy = { {2.1, 0.0}, {4.1, 0.0}, {6.1, 0.0}, {11.0, 0.0} };
		using blas::operators::operator-=;
		yy -= x;
		yy -= y;
		using blas::operators::norm;
		BOOST_REQUIRE( norm(yy) == 0 );
		using blas::operators::operator==;
		BOOST_REQUIRE( operator==(yy, 0) );
		BOOST_REQUIRE( yy == 0 );
	}
}

BOOST_AUTO_TEST_CASE(cublas_axpy_complex_alpha) {
	namespace blas = multi::blas;
	complex const I{0.0, 1.0};

	using T = complex;
	using Alloc =  thrust::cuda::allocator<complex>;

	multi::array<complex, 1, Alloc> const x = { {1.1, 0.0}, {2.1, 0.0}, {3.1, 0.0}, {4.1, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 1, Alloc> y = { {2.1, 0.0}, {4.1, 0.0}, {6.1, 0.0}, {11.0, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming

	blas::axpy(3.0, x, y);
	std::cout << y[0] << std::endl;
	BOOST_REQUIRE( static_cast<complex>(y[0]) == 5.4 + I*0.0 );
	{
		multi::array<complex, 1, Alloc> yy = { {2.1, 0.0}, {4.1, 0.0}, {6.1, 0.0}, {11.0, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
		thrust::transform(x.begin(), x.end(), yy.begin(), yy.begin(), [aa=3.0] __device__ (T ex, T ey) {return aa*ex + ey;});
		BOOST_TEST( yy == y , boost::test_tools::per_element() );
	}
	{
		multi::array<complex, 1, Alloc> yy = { {2.1, 0.0}, {4.1, 0.0}, {6.1, 0.0}, {11.0, 0.0} };
		using blas::operators::operator+=;
		using blas::operators::operator*;
		yy += 3.0*x;
		BOOST_REQUIRE( yy == y );
	}
}

BOOST_AUTO_TEST_CASE(cublas_one_gemv_conj_complex_zero) {
	namespace blas = multi::blas;
	using T = complex;
	complex const I{0.0, 1.0};
	using Alloc =  thrust::cuda::allocator<complex>;

	// NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 2, Alloc> const A = {
		{ { 9.0, 0.0}, {24.0, 0.0}, {30.0, 0.0}, {9.0, 0.0} },
		{ { 4.0, 0.0}, {10.0, 0.0}, {12.0, 0.0}, {7.0, 0.0} },
		{ {14.0, 0.0}, {16.0, 0.0}, {36.0, 0.0}, {1.0, 0.0} },
	};
	multi::array<complex, 1, Alloc> const x = { {1.1, 0.0}, {2.1, 0.0}, {3.1, 0.0}, {4.1, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 1, Alloc> y = { {1.1, 0.0}, {2.1, 0.0}, {3.1, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
	blas::gemv(1.0, A, x, 0.0, y);
	{

		multi::array<complex, 1, Alloc> yy = { {1.1, 0.0}, {2.1, 0.0}, {3.1, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
		std::transform(begin(A), end(A), begin(yy), [&x] (auto const& Ac) {return blas::dot(Ac, x);});

		BOOST_REQUIRE( static_cast<complex>(y[0]) == static_cast<complex>(yy[0]) );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );
	}
	{
		multi::array<complex, 1, Alloc> yy = { {1.1, 0.0}, {2.1, 0.0}, {3.1, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
		yy = blas::gemv(1.0, A, x);
		BOOST_REQUIRE( static_cast<complex>(y[0]) == static_cast<complex>(yy[0]) );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );
	}
	{
		multi::array<complex, 1, Alloc> yy = blas::gemv(1.0, A, x);
		BOOST_REQUIRE( static_cast<complex>(y[0]) == static_cast<complex>(yy[0]) );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );

	}
	{
		using blas::operators::operator%;

		multi::array<complex, 1, Alloc> yy = { {1.1, 0.0}, {2.1, 0.0}, {3.1, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
		yy = A % x;
		BOOST_REQUIRE( static_cast<complex>(y[0]) == static_cast<complex>(yy[0]) );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );
	}
}

BOOST_AUTO_TEST_CASE(cublas_one_gemv_complex_conj_zero) {
	namespace blas = multi::blas;
	using T = complex;
	using Alloc =  thrust::cuda::allocator<complex>;
	complex const I{0.0, 1.0};

	// NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 2, Alloc> const A = {
		{  9.0 + I*0.0, 24.0 + I* 0.0, 30.0 + I* 0.0, 9.0 + I* 0.0 },
		{  4.0 + I*0.0, 10.0 + I* 0.0, 12.0 + I* 0.0, 7.0 + I* 0.0 },
		{ 14.0 + I*0.0, 16.0 + I* 0.0, 36.0 + I* 0.0, 1.0 + I* 0.0 },
	};
	multi::array<complex, 1, Alloc> const x = { 1.1 + I* 0.0, 2.1 + I* 0.0, 3.1 + I* 0.0};  // NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 1, Alloc> y = { 1.1 + I* 0.0, 2.1 +I* 0.0, 3.1 + I* 0.0, 6.7 + I*0.0 };  // NOLINT(readability-identifier-length) BLAS naming
	blas::gemv(1.0, blas::T(A), x, 0.0, y);
	{
		multi::array<complex, 1, Alloc> yy = { 1.1 + I* 0.0, 2.1 +I* 0.0, 3.1 + I* 0.0, 6.7 + I*0.0 };  // NOLINT(readability-identifier-length) BLAS naming
		using blas::operators::operator*;
		std::transform(begin(transposed(A)), end(transposed(A)), begin(yy), [&x] (auto const& Ac) {return blas::dot(Ac, x);});

		BOOST_REQUIRE_CLOSE( static_cast<complex>(y[0]).real(), static_cast<complex>(yy[0]).real(), 1e-7 );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );
	}
	{
		multi::array<complex, 1, Alloc> yy = { 1.1 + I* 0.0, 2.1 +I* 0.0, 3.1 + I* 0.0, 6.7 + I*0.0 };  // NOLINT(readability-identifier-length) BLAS naming
		yy = blas::gemv(1.0, blas::T(A), x);
		BOOST_REQUIRE( static_cast<complex>(y[0]) == static_cast<complex>(yy[0]) );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );
	}
	{
		multi::array<complex, 1, Alloc> yy = blas::gemv(1.0, blas::T(A), x);
		BOOST_REQUIRE( static_cast<complex>(y[0]) == static_cast<complex>(yy[0]) );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );
	}
	{
		using blas::operators::operator%;

		multi::array<complex, 1, Alloc> yy = { 1.1 + I* 0.0, 2.1 +I* 0.0, 3.1 + I* 0.0, 6.7 + I*0.0 };  // NOLINT(readability-identifier-length) BLAS naming
		yy = ~A % x;
		BOOST_REQUIRE( static_cast<complex>(y[0]) == static_cast<complex>(yy[0]) );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );
	}
}

template<class T> void what(T&&) = delete;

BOOST_AUTO_TEST_CASE(cublas_one_gemv_complex_zero) {
	namespace blas = multi::blas;
	using T = complex;
	complex const I{0.0, 1.0};
	using Alloc =  thrust::cuda::allocator<complex>;

	// NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 2, Alloc> const A = {
		{ { 9.0, 0.0}, {24.0, 0.0}, {30.0, 0.0}, {9.0, 0.0} },
		{ { 4.0, 0.0}, {10.0, 0.0}, {12.0, 0.0}, {7.0, 0.0} },
		{ {14.0, 0.0}, {16.0, 0.0}, {36.0, 0.0}, {1.0, 0.0} },
	};
	multi::array<complex, 1, Alloc> const x = { {1.1, 0.0}, {2.1, 0.0}, {3.1, 0.0}, {4.1, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 1, Alloc> y = { {1.1, 0.0}, {2.1, 0.0}, {3.1, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
	blas::gemv(1.0, blas::J(A), x, 0.0, y);
	{
		multi::array<complex, 1, Alloc> yy = { {1.1, 0.0}, {2.1, 0.0}, {3.1, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
		std::transform(begin(A), end(A), begin(yy), [&x] (auto const& Ac) {
			using blas::operators::operator*;  // nvcc 11.8 needs this to be inside lambda
			return blas::dot(*Ac, x);}
		);

		BOOST_REQUIRE( static_cast<complex>(y[0]) == static_cast<complex>(yy[0]) );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );
	}
	{
		multi::array<complex, 1, Alloc> yy = { {1.1, 0.0}, {2.1, 0.0}, {3.1, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
		yy = blas::gemv(1.0, blas::J(A), x);
		BOOST_REQUIRE( static_cast<complex>(y[0]) == static_cast<complex>(yy[0]) );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );
	}
	{
		multi::array<complex, 1, Alloc> yy = blas::gemv(1.0, blas::J(A), x);
		BOOST_REQUIRE( static_cast<complex>(y[0]) == static_cast<complex>(yy[0]) );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );

	}
	{
		using blas::operators::operator%;
		using blas::operators::operator*;

		multi::array<complex, 1, Alloc> yy = { {1.1, 0.0}, {2.1, 0.0}, {3.1, 0.0} };  // NOLINT(readability-identifier-length) BLAS naming
		yy = *A % x;
		BOOST_REQUIRE( static_cast<complex>(y[0]) == static_cast<complex>(yy[0]) );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );
	}
}


BOOST_AUTO_TEST_CASE(cublas_one_gemv_complex_conjtrans_zero) {
	namespace blas = multi::blas;
	using T = complex;
	using Alloc =  std::allocator<complex>;  // thrust::cuda::allocator<complex>;
	complex const I{0.0, 1.0};

	// NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 2, Alloc> const A = {
		{  9.0 + I*0.0, 24.0 + I* 0.0, 30.0 + I* 0.0, 9.0 + I* 0.0 },
		{  4.0 + I*0.0, 10.0 + I* 0.0, 12.0 + I* 0.0, 7.0 + I* 0.0 },
		{ 14.0 + I*0.0, 16.0 + I* 0.0, 36.0 + I* 0.0, 1.0 + I* 0.0 },
	};
	multi::array<complex, 1, Alloc> const x = { 1.1 + I* 0.0, 2.1 + I* 0.0, 3.1 + I* 0.0};  // NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 1, Alloc> y = { 1.1 + I* 0.0, 2.1 +I* 0.0, 3.1 + I* 0.0, 6.7 + I*0.0 };  // NOLINT(readability-identifier-length) BLAS naming

	// blas::gemv(1.0, blas::H(A), x, 0.0, y);

	{
		multi::array<complex, 1, Alloc> yy = { 1.1 + I* 0.0, 2.1 +I* 0.0, 3.1 + I* 0.0, 6.7 + I*0.0 };  // NOLINT(readability-identifier-length) BLAS naming
		std::transform(begin(transposed(A)), end(transposed(A)), begin(yy), [&x] (auto const& Ac) {
			using blas::operators::operator*;  // nvcc 11.8 needs this to be inside lambda
			return blas::dot(*Ac, x);}
		);

		BOOST_REQUIRE_CLOSE( static_cast<complex>(yy[0]).real() ,  61.7, 1.e-7  );
		BOOST_REQUIRE_CLOSE( static_cast<complex>(yy[1]).real() ,  97.0, 1.e-7  );
		BOOST_REQUIRE_CLOSE( static_cast<complex>(yy[2]).real() , 169.8, 1.e-7  );
		BOOST_REQUIRE_CLOSE( static_cast<complex>(yy[3]).real() ,  27.7, 1.e-7  );

		using blas::operators::operator*;
		BOOST_REQUIRE_CLOSE( static_cast<complex>(yy[0]).real() , (+blas::dot(*(~A)[0], x)).real() , 1.e-7  );
		BOOST_REQUIRE_CLOSE( static_cast<complex>(yy[1]).real() , (+blas::dot(*(~A)[1], x)).real() , 1.e-7  );
	}
}

BOOST_AUTO_TEST_CASE(cublas_one_gemv_complex_trans_one) {
	namespace blas = multi::blas;
	using T = complex;
	using Alloc =  thrust::cuda::allocator<complex>;
	complex const I{0.0, 1.0};

	// NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 2, Alloc> const A = {
		{  9.0 + I*0.0, 24.0 + I* 0.0, 30.0 + I* 0.0, 9.0 + I* 0.0 },
		{  4.0 + I*0.0, 10.0 + I* 0.0, 12.0 + I* 0.0, 7.0 + I* 0.0 },
		{ 14.0 + I*0.0, 16.0 + I* 0.0, 36.0 + I* 0.0, 1.0 + I* 0.0 },
	};
	multi::array<complex, 1, Alloc> const x = { 1.1 + I* 0.0, 2.1 + I* 0.0, 3.1 + I* 0.0};  // NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 1, Alloc> y = { 1.1 + I* 0.0, 2.1 +I* 0.0, 3.1 + I* 0.0, 6.7 + I*0.0 };  // NOLINT(readability-identifier-length) BLAS naming
	blas::gemv(3.0 + I*4.0, blas::T(A), x, 1.0, y);
	{
		multi::array<complex, 1, Alloc> yy = { 1.1 + I* 0.0, 2.1 +I* 0.0, 3.1 + I* 0.0, 6.7 + I*0.0 };  // NOLINT(readability-identifier-length) BLAS naming
		// using blas::operators::operator*;
		std::transform(begin(transposed(A)), end(transposed(A)), begin(yy), begin(yy), [&x,aa=3.0 + I*4.0,bb=1.0] (auto const& Ac, complex e) {return aa*blas::dot(Ac, x) + bb*e;});

		BOOST_REQUIRE_CLOSE( static_cast<complex>(y[0]).real(), static_cast<complex>(yy[0]).real(), 1e-7 );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );
	}
	{
		multi::array<complex, 1, Alloc> yy = { 1.1 + I* 0.0, 2.1 +I* 0.0, 3.1 + I* 0.0, 6.7 + I*0.0 };  // NOLINT(readability-identifier-length) BLAS naming
		yy += blas::gemv(3.0 + I*4.0, blas::T(A), x);

		BOOST_REQUIRE_CLOSE( static_cast<complex>(y[0]).real(), static_cast<complex>(yy[0]).real(), 1e-7 );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );
	}
	{
		multi::array<complex, 1, Alloc> yy = { 1.1 + I* 0.0, 2.1 +I* 0.0, 3.1 + I* 0.0, 6.7 + I*0.0 };  // NOLINT(readability-identifier-length) BLAS naming
		using blas::operators::operator*;
		yy += (3.0 + I*4.0)* ~A % x;

		BOOST_REQUIRE_CLOSE( static_cast<complex>(y[0]).real(), static_cast<complex>(yy[0]).real(), 1e-7 );
		BOOST_REQUIRE( static_cast<complex>(y[1]) == static_cast<complex>(yy[1]) );
		BOOST_REQUIRE( static_cast<complex>(y[2]) == static_cast<complex>(yy[2]) );
	}
}

BOOST_AUTO_TEST_CASE(cublas_one_gemm_complex_trans_none) {
	namespace blas = multi::blas;
	using T = complex;
	using Alloc =  thrust::cuda::allocator<complex>;
	complex const I{0.0, 1.0};

	// NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 2, Alloc> const A = {
		{1.0 - 2.0 * I, 9.0 - 1.0 * I},
		{2.0 + 3.0 * I, 1.0 - 2.0 * I},
	};
	multi::array<complex, 2, Alloc> const B = {
		{3.0 - 4.0 * I, 19.0 - 1.0 * I},
		{1.0 + 5.0 * I,  8.0 - 8.0 * I},
	};
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		blas::gemm({1.0, 0.0}, A, B, {0.0, 0.0}, C);

		// std::transform(begin(transposed(B)), end(transposed(B)), begin(transposed(C_copy)), begin(transposed(C_copy)),
		//  [&A, aa=1.0, bb=0.0] (auto const& Bc, auto&& Cc) {return blas::gemv(aa, A, Bc, bb, std::move(Cc));}
		// );
		std::transform(begin(A), end(A), begin(C_copy), end(C_copy),
			[&B, aa=1.0, bb=0] (auto const& Ar, auto&& Cr) {return blas::gemv(aa, blas::T(B), Ar, bb, std::move(Cr));}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		C = blas::gemm(1.0 + I*0.0, A, B);

		// std::transform(begin(transposed(B)), end(transposed(B)), begin(transposed(C_copy)), begin(transposed(C_copy)),
		//  [&A, aa=1.0, bb=0.0] (auto const& Bc, auto&& Cc) {return blas::gemv(aa, A, Bc, bb, std::move(Cc));}
		// );
		std::transform(begin(A), end(A), begin(C_copy), begin(C_copy), [&B, aa=1.0, bb=0.0] (auto const& Ar, auto&& Cr) {
			return blas::gemv(aa, blas::T(B), Ar, bb, std::move(Cr));
		}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		C += blas::gemm(1.0 + I*0.0, A, B);

		std::transform(begin(transposed(B)), end(transposed(B)), begin(transposed(C_copy)), begin(transposed(C_copy)),
			[&A, aa=1.0, bb=1.0] (auto const& Bc, auto&& Cc) {return blas::gemv(aa, A, Bc, bb, std::move(Cc));}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		using blas::operators::operator*;
		using blas::operators::operator+=;
		C += A*B;

		std::transform(begin(A), end(A), begin(C_copy), begin(C_copy), [&B, aa=1.0, bb=1.0] (auto const& Ar, auto&& Cr) {
			return blas::gemv(aa, blas::T(B), Ar, bb, std::move(Cr));
		}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
}

BOOST_AUTO_TEST_CASE(cublas_one_gemm_complex_trans_second) {
	namespace blas = multi::blas;
	using T = complex;
	using Alloc =  thrust::cuda::allocator<complex>;
	complex const I{0.0, 1.0};

	// NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 2, Alloc> const A = {
		{1.0 - 2.0 * I, 9.0 - 1.0 * I},
		{2.0 + 3.0 * I, 1.0 - 2.0 * I},
	};
	multi::array<complex, 2, Alloc> const B = {
		{3.0 - 4.0 * I, 19.0 - 1.0 * I},
		{1.0 + 5.0 * I,  8.0 - 8.0 * I},
	};
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		blas::gemm({1.0, 0.0}, A, blas::T(B), {0.0, 0.0}, C);

		std::transform(begin(B), end(B), begin(transposed(C_copy)), begin(transposed(C_copy)),
			[&A, aa=1.0, bb=0.0] (auto const& Bc, auto&& Cc) {return blas::gemv(aa, A, Bc, bb, std::move(Cc));}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		C = blas::gemm(1.0 + I*0.0, A, blas::T(B));

		// std::transform(begin(transposed(B)), end(transposed(B)), begin(transposed(C_copy)), begin(transposed(C_copy)),
		//  [&A, aa=1.0, bb=0.0] (auto const& Bc, auto&& Cc) {return blas::gemv(aa, A, Bc, bb, std::move(Cc));}
		// );
		std::transform(begin(A), end(A), begin(C_copy), begin(C_copy), [&B, aa=1.0, bb=0.0] (auto const& Ac, auto&& Cr) {
			return blas::gemv(aa, B, Ac, bb, std::move(Cr));
		});

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		C += blas::gemm(1.0 + I*0.0, A, blas::T(B));

		std::transform(begin(B), end(B), begin(transposed(C_copy)), begin(transposed(C_copy)),
			[&A, aa=1.0, bb=1.0] (auto const& Bc, auto&& Cc) {return blas::gemv(aa, A, Bc, bb, std::move(Cc));}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		using blas::operators::operator*;
		using blas::operators::operator+=;
		C += A * ~B;

		std::transform(begin(A), end(A), begin(C_copy), begin(C_copy), [&B, aa=1.0, bb=1.0] (auto const& Ar, auto&& Cr) {
			return blas::gemv(aa, B, Ar, bb, std::move(Cr));
		}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		using blas::operators::operator*;
		using blas::operators::operator+=;
		C += 2.0*(A * ~B);

		std::transform(begin(A), end(A), begin(C_copy), begin(C_copy), [&B, aa=2.0, bb=1.0] (auto const& Ar, auto&& Cr) {
			return blas::gemv(aa, B, Ar, bb, std::move(Cr));
		}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
}

BOOST_AUTO_TEST_CASE(cublas_one_gemm_complex_trans_first) {
	namespace blas = multi::blas;
	using T = complex;
	using Alloc =  thrust::cuda::allocator<complex>;
	complex const I{0.0, 1.0};

	// NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 2, Alloc> const A = {
		{1.0 - 2.0 * I, 9.0 - 1.0 * I},
		{2.0 + 3.0 * I, 1.0 - 2.0 * I},
	};
	multi::array<complex, 2, Alloc> const B = {
		{3.0 - 4.0 * I, 19.0 - 1.0 * I},
		{1.0 + 5.0 * I,  8.0 - 8.0 * I},
	};
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		blas::gemm({1.0, 0.0}, blas::T(A), B, {0.0, 0.0}, C);

		std::transform(begin(transposed(B)), end(transposed(B)), begin(transposed(C_copy)), begin(transposed(C_copy)),
			[&A, aa=1.0, bb=0.0] (auto const& Bc, auto&& Cc) {return blas::gemv(aa, blas::T(A), Bc, bb, std::move(Cc));}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		C = blas::gemm(1.0 + I*0.0, blas::T(A), B);

		// std::transform(begin(transposed(B)), end(transposed(B)), begin(transposed(C_copy)), begin(transposed(C_copy)),
		//  [&A, aa=1.0, bb=0.0] (auto const& Bc, auto&& Cc) {return blas::gemv(aa, A, Bc, bb, std::move(Cc));}
		// );
		std::transform(begin(transposed(A)), end(transposed(A)), begin(C_copy), begin(C_copy), [&B, aa=1.0, bb=0.0] (auto const& Ac, auto&& Cr) {
			return blas::gemv(aa, blas::T(B), Ac, bb, std::move(Cr));
		});

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		C += blas::gemm(1.0 + I*0.0, blas::T(A), B);

		std::transform(begin(transposed(B)), end(transposed(B)), begin(transposed(C_copy)), begin(transposed(C_copy)),
			[&A, aa=1.0, bb=1.0] (auto const& Bc, auto&& Cc) {return blas::gemv(aa, blas::T(A), Bc, bb, std::move(Cc));}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		using blas::operators::operator*;
		using blas::operators::operator+=;
		C += ~A * B;

		std::transform(begin(transposed(A)), end(transposed(A)), begin(C_copy), begin(C_copy), [&B, aa=1.0, bb=1.0] (auto const& Ar, auto&& Cr) {
			return blas::gemv(aa, blas::T(B), Ar, bb, std::move(Cr));
		}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		using blas::operators::operator*;
		using blas::operators::operator+=;
		C += 2.0*(~A * B);

		std::transform(begin(transposed(A)), end(transposed(A)), begin(C_copy), begin(C_copy), [&B, aa=2.0, bb=1.0] (auto const& Ar, auto&& Cr) {
			return blas::gemv(aa, blas::T(B), Ar, bb, std::move(Cr));
		}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
}

BOOST_AUTO_TEST_CASE(cublas_one_gemm_complex_trans_both) {
	namespace blas = multi::blas;
	using T = complex;
	using Alloc =  thrust::cuda::allocator<complex>;
	complex const I{0.0, 1.0};

	// NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 2, Alloc> const A = {
		{1.0 - 2.0 * I, 9.0 - 1.0 * I},
		{2.0 + 3.0 * I, 1.0 - 2.0 * I},
	};
	multi::array<complex, 2, Alloc> const B = {
		{3.0 - 4.0 * I, 19.0 - 1.0 * I},
		{1.0 + 5.0 * I,  8.0 - 8.0 * I},
	};
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		blas::gemm({1.0, 0.0}, blas::T(A), blas::T(B), {0.0, 0.0}, C);

		std::transform(begin(B), end(B), begin(transposed(C_copy)), begin(transposed(C_copy)),
			[&A, aa=1.0, bb=0.0] (auto const& Br, auto&& Cc) {return blas::gemv(aa, blas::T(A), Br, bb, std::move(Cc));}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		C = blas::gemm(1.0 + I*0.0, blas::T(A), blas::T(B));

		// std::transform(begin(transposed(B)), end(transposed(B)), begin(transposed(C_copy)), begin(transposed(C_copy)),
		//  [&A, aa=1.0, bb=0.0] (auto const& Bc, auto&& Cc) {return blas::gemv(aa, A, Bc, bb, std::move(Cc));}
		// );
		std::transform(begin(transposed(A)), end(transposed(A)), begin(C_copy), begin(C_copy), [&B, aa=1.0, bb=0.0] (auto const& Ac, auto&& Cr) {
			return blas::gemv(aa, B, Ac, bb, std::move(Cr));
		});

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		C += blas::gemm(1.0 + I*0.0, blas::T(A), blas::T(B));

		std::transform(begin(B), end(B), begin(transposed(C_copy)), begin(transposed(C_copy)),
			[&A, aa=1.0, bb=1.0] (auto const& Br, auto&& Cc) {return blas::gemv(aa, blas::T(A), Br, bb, std::move(Cc));}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		using blas::operators::operator*;
		using blas::operators::operator+=;
		C += ~A * ~B;

		std::transform(begin(transposed(A)), end(transposed(A)), begin(C_copy), begin(C_copy), [&B, aa=1.0, bb=1.0] (auto const& Ar, auto&& Cr) {
			return blas::gemv(aa, B, Ar, bb, std::move(Cr));
		}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto C_copy = C;
		using blas::operators::operator*;
		using blas::operators::operator+=;
		C += 2.0*(~A * ~B);

		std::transform(begin(transposed(A)), end(transposed(A)), begin(C_copy), begin(C_copy), [&B, aa=2.0, bb=1.0] (auto const& Ar, auto&& Cr) {
			return blas::gemv(aa, B, Ar, bb, std::move(Cr));
		}
		);

		BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
		BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	}
}

BOOST_AUTO_TEST_CASE(cublas_one_gemm_complex_conj_second) {
	namespace blas = multi::blas;
	using T = complex;
	using Alloc =  std::allocator<complex>;  // thrust::cuda::allocator<complex>;
	complex const I{0.0, 1.0};

	// NOLINT(readability-identifier-length) BLAS naming
	multi::array<complex, 2, Alloc> const A = {
		{1.0 - 2.0 * I, 9.0 - 1.0 * I},
		{2.0 + 3.0 * I, 1.0 - 2.0 * I},
	};
	multi::array<complex, 2, Alloc> const B = {
		{3.0 - 4.0 * I, 19.0 - 1.0 * I},
		{1.0 + 5.0 * I,  8.0 - 8.0 * I},
	};
	{
		multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
		auto CC = C;
		auto C_copy = CC;
		// blas::gemm({1.0, 0.0}, A, blas::J(B), {0.0, 0.0}, C);
		blas::gemm({1.0, 0.0}, blas::T(B), blas::H(A), {0.0, 0.0}, C_copy);
		{
			auto const [is, js] = C.extensions();
			for(auto i : is) {
				for(auto j : js) {
					C[i][j] *= 0.0;
					for(auto k : B.extension()) {
						C[i][j] += A[i][k]*conj(B[k][j]);
					}
				}
			}
		}
		{
			std::transform(begin(A), end(A), begin(CC), begin(CC), [BT = transposed(B)](auto const& Ar, auto&& Cr) {
				return std::transform(
					begin(BT), end(BT), begin(Cr), begin(Cr), [&Ar](auto const& BCr, auto&& Ce) {
						return 1.0*blas::dot(Ar, blas::C(BCr)) + 0.0*Ce;
					}
				), std::move(Cr);
			});
		}
		BOOST_TEST_REQUIRE( static_cast<complex>(CC[1][0]).real() == static_cast<complex>(C[1][0]).real() );
		BOOST_TEST_REQUIRE( static_cast<complex>(CC[1][0]).imag() == static_cast<complex>(C[1][0]).imag() );

		BOOST_TEST_REQUIRE( static_cast<complex>(CC[0][1]).real() == static_cast<complex>(C[0][1]).real() );
		BOOST_TEST_REQUIRE( static_cast<complex>(CC[0][1]).imag() == static_cast<complex>(C[0][1]).imag() );

		BOOST_TEST_REQUIRE( static_cast<complex>(C_copy[1][0]).real() == +static_cast<complex>(C[0][1]).real() );
		BOOST_TEST_REQUIRE( static_cast<complex>(C_copy[1][0]).imag() == -static_cast<complex>(C[0][1]).imag() );
	}
	//  auto const stl_gemvH = [](auto aa, auto const& A, auto const& x, auto bb, auto&& yy) {
	//      std::transform(begin(transposed(A)), end(transposed(A)), begin(yy), begin(yy), [aa, &x, bb] (auto const& Ac, auto&& ye) {
	//          using blas::operators::operator*;  // nvcc 11.8 needs this to be inside lambda
	//          return aa*static_cast<complex>(blas::dot(*Ac, x)) + bb*ye;}
	//      );
	//      return std::move(yy);
	//  };
	//  auto const stl_gemv = [](auto aa, auto const& A, auto const& x, auto bb, auto&& yy) {
	//      std::transform(begin(A), end(A), begin(yy), begin(yy), [aa, &x, bb] (auto const& Ar, auto&& ye) {
	//          using blas::operators::operator*;  // nvcc 11.8 needs this to be inside lambda
	//          return aa*static_cast<complex>(blas::dot(Ar, x)) + bb*ye;}
	//      );
	//      return std::move(yy);
	//  };

	//  std::transform(begin(A), end(A), begin(C), end(C),
	//      [&B, aa=1.0, bb=0, stl_gemv] (auto const& Ar, auto&& Cr) {return stl_gemv(aa, blas::H(B), Ar, bb, std::move(Cr));}
	//  );

	//  std::transform(
	//      begin(A), end(A), begin(C), end(C),
	//      [&B, aa=1.0, bb=0, stl_gemv] (auto const& Ar, auto&& Cr) {
	//          std::transform(begin(blas::H(B)), end(blas::H(B)), begin(Cr), begin(Cr), [aa, &Ar, bb] (auto const& Bc, auto&& ye) {
	//              using blas::operators::operator*;  // nvcc 11.8 needs this to be inside lambda
	//              return aa*static_cast<complex>(blas::dot(Bc, Ar)) + bb*ye;
	//          });
	//          return std::move(Cr);
	//      }
	//  );

	//  BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
	//  BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	// }
	// {
	//  multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
	//  auto C_copy = C;
	//  C = blas::gemm(1.0 + I*0.0, A, blas::T(B));

	//  // std::transform(begin(transposed(B)), end(transposed(B)), begin(transposed(C_copy)), begin(transposed(C_copy)),
	//  //  [&A, aa=1.0, bb=0.0] (auto const& Bc, auto&& Cc) {return blas::gemv(aa, A, Bc, bb, std::move(Cc));}
	//  // );
	//  std::transform(begin(A), end(A), begin(C_copy), begin(C_copy), [&B, aa=1.0, bb=0.0] (auto const& Ac, auto&& Cr) {
	//      return blas::gemv(aa, B, Ac, bb, std::move(Cr));
	//  });

	//  BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
	//  BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	// }
	// {
	//  multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
	//  auto C_copy = C;
	//  C += blas::gemm(1.0 + I*0.0, A, blas::T(B));

	//  std::transform(begin(B), end(B), begin(transposed(C_copy)), begin(transposed(C_copy)),
	//      [&A, aa=1.0, bb=1.0] (auto const& Bc, auto&& Cc) {return blas::gemv(aa, A, Bc, bb, std::move(Cc));}
	//  );

	//  BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
	//  BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	// }
	// {
	//  multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
	//  auto C_copy = C;
	//  using blas::operators::operator*;
	//  using blas::operators::operator+=;
	//  C += A * ~B;

	//  std::transform(begin(A), end(A), begin(C_copy), begin(C_copy), [&B, aa=1.0, bb=1.0] (auto const& Ar, auto&& Cr) {
	//      return blas::gemv(aa, B, Ar, bb, std::move(Cr));
	//  }
	//  );

	//  BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
	//  BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	// }
	// {
	//  multi::array<complex, 2, Alloc> C({2, 2}, {3.0, 0.0});  // NOLINT(readability-identifier-length) conventional BLAS naming
	//  auto C_copy = C;
	//  using blas::operators::operator*;
	//  using blas::operators::operator+=;
	//  C += 2.0*(A * ~B);

	//  std::transform(begin(A), end(A), begin(C_copy), begin(C_copy), [&B, aa=2.0, bb=1.0] (auto const& Ar, auto&& Cr) {
	//      return blas::gemv(aa, B, Ar, bb, std::move(Cr));
	//  }
	//  );

	//  BOOST_REQUIRE( static_cast<complex>(C_copy[1][0]) == static_cast<complex>(C[1][0]) );
	//  BOOST_REQUIRE( static_cast<complex>(C_copy[0][1]) == static_cast<complex>(C[0][1]) );
	// }
}

#endif
